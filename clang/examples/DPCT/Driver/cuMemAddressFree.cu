// Option: --use-experimental-features=virtual_mem
#include <hip/hip_runtime.h>
void test(hipDeviceptr_t ptr, size_t size) {
  // Start
  hipMemAddressFree(ptr /*CUdeviceptr*/, size /*size_t*/);
  // End
}