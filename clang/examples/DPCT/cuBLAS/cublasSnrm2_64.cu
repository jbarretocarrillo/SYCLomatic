
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, const float *x, int64_t incx,
          float *res) {
  // Start
  hipblasSnrm2_64(handle /*cublasHandle_t*/, n /*int64_t*/, x /*const float **/,
                 incx /*int64_t*/, res /*float **/);
  // End
}
